#include "hip/hip_runtime.h"

#include <memetico/gpu/cuda.cuh>

using namespace std;
using namespace cusr;

namespace cusr {
    
    void copySubset(GPUDataset *dataset_struct, vector<size_t> &idxs) {

        // copy label set
        size_t *device_idx_arr;
        hipMalloc((void **) &device_idx_arr, sizeof(size_t) * idxs.size());
        hipMemcpy(device_idx_arr, thrust::raw_pointer_cast(idxs.data()), sizeof(size_t) * idxs.size(), hipMemcpyHostToDevice);
        dataset_struct->subset = device_idx_arr;
        dataset_struct->subset_size = idxs.size();

    }

    void freeSubset(GPUDataset *dataset_struct) {
        hipFree(dataset_struct->subset);
        dataset_struct->subset_size = 0;
    }

    void copyDatasetAndLabel(GPUDataset *dataset_struct, vector<vector<float>> &dataset, vector<float> &label) {
        dataset_struct->dataset_size = dataset.size();

        // format dataset into column-major
        int data_size = dataset.size();
        int variable_num = dataset[0].size();

        vector<float> device_dataset;

        for (int i = 0; i < variable_num; i++) {
            for (int j = 0; j < data_size; j++) {
                device_dataset.emplace_back(dataset[j][i]);
            }
        }

        // copy dataset
        float *device_dataset_arr;
        size_t dataset_pitch;
        auto errr0 = hipMallocPitch((void **) &device_dataset_arr, &dataset_pitch, sizeof(float) * data_size, variable_num);
        auto errr1 = hipMemcpy2D(device_dataset_arr, dataset_pitch, thrust::raw_pointer_cast(device_dataset.data()),
                        sizeof(float) * data_size, sizeof(float) * data_size, variable_num, hipMemcpyHostToDevice);

        // Check dataset
        //float *device_return_arr;
        //auto err1 = hipMemcpy2D(device_return_arr, sizeof(float) * data_size, device_dataset_arr, dataset_pitch, sizeof(float) * data_size, variable_num, hipMemcpyDeviceToHost);
        //for (int i = 0; i < data_size*variable_num; i++) {
        //    cout << device_return_arr[i] << endl;
        //}

        dataset_struct->dataset_pitch = dataset_pitch;
        dataset_struct->dataset = device_dataset_arr;

        // copy label set
        float *device_label_arr;
        auto err0 = hipMalloc((void **) &device_label_arr, sizeof(float) * data_size);
        auto err1 = hipMemcpy(device_label_arr, thrust::raw_pointer_cast(label.data()), sizeof(float) * data_size, hipMemcpyHostToDevice);

        // Check dataset
        //float *device_return_label_arr;
        //auto err2 = hipMemcpy(device_return_label_arr, device_label_arr, sizeof(float)*data_size, hipMemcpyDeviceToHost);
        //for (int i = 0; i < data_size; i++) {
        //    cout << device_return_label_arr[i] << endl;
        //}

        dataset_struct->label = device_label_arr;

    }

    void freeDataSetAndLabel(GPUDataset *dataset_struct) {
        hipFree(dataset_struct->dataset);
        hipFree(dataset_struct->label);
    }

    __constant__ float d_nodeValue[MAX_PREFIX_LEN];
    __constant__ float d_nodeType[MAX_PREFIX_LEN];

#define S_OFF THREAD_PER_BLOCK * (CUSR_DEPTH + 1) * blockIdx.x + top * THREAD_PER_BLOCK + threadIdx.x

    __global__ void
    calFitnessGPU_MSE(int len, float *ds, int dsPitch, float *label, size_t *idxs, float *stack, float *result,
                        int dataset_size, bool is_subset, bool do_print) {

        extern __shared__ float shared[];
        shared[threadIdx.x] = 0;
        
        // each thread is responsible for one datapoint
        int dataset_no = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

        //if(do_print)
        //printf("Starting calFitnessGPU_MSE ds_size: %d is_subset: %d \n", dataset_no, is_subset);

        if (dataset_no < dataset_size) {

            int top = 0;

            // do stack operation according to the type of each node
            for (int i = len - 1; i >= 0; i--) {

                int node_type = d_nodeType[i];
                float node_value = d_nodeValue[i];

                if (node_type == NodeType::CONST) {
                    stack[S_OFF] = node_value;
                    top++;
                } else if (node_type == NodeType::VAR) {
                    int var_num = node_value;

                    if( is_subset) {
                        stack[S_OFF] = ((float *) ((char *) ds + var_num * dsPitch))[idxs[dataset_no]];
                        //if( dataset_no == 1 && do_print)
                        //    printf("S_OFF: %d, Node: Var, Num: %d, Stack: %f \n", S_OFF, var_num, stack[S_OFF]);
                    } else {
                        stack[S_OFF] = ((float *) ((char *) ds + var_num * dsPitch))[dataset_no];
                        //if( dataset_no == 1 && do_print)
                        //    printf("S_OFF: %d, Node: Var, Num: %d, Stack: %f \n", S_OFF, var_num, stack[S_OFF]);
                    }
                    top++;

                } else if (node_type == NodeType::UFUNC) {
                    int function = node_value;
                    top--;
                    float var1 = stack[S_OFF];
                    if (function == Function::SIN) {
                        stack[S_OFF] = std::sin(var1);
                        top++;
                    } else if (function == Function::COS) {
                        stack[S_OFF] = std::cos(var1);
                        top++;
                    } else if (function == Function::TAN) {
                        stack[S_OFF] = std::tan(var1);
                        top++;
                    } else if (function == Function::LOG) {
                        if (var1 <= 0) {
                            stack[S_OFF] = -1.0f;
                            top++;
                        } else {
                            stack[S_OFF] = std::log(var1);
                            top++;
                        }
                    } else if (function == Function::INV) {
                        if (var1 == 0) {
                            var1 = DELTA;
                        }
                        stack[S_OFF] = 1.0f / var1;
                        top++;
                    }
                } else // if (node_type == NodeType::BFUNC)
                {
                    int function = node_value;
                    top--;
                    float var1 = stack[S_OFF];
                    top--;
                    float var2 = stack[S_OFF];

                    if (function == Function::ADD) {
                        stack[S_OFF] = var1 + var2;
                        top++;
                        //if( dataset_no == 1 && do_print)
                        //    printf("S_OFF: %d, Node: Add, Nums: '%f+%f', Stack: %f \n", S_OFF, var1, var2, stack[S_OFF]);
                    } else if (function == Function::SUB) {
                        stack[S_OFF] = var1 - var2;
                        top++;
                        //if( dataset_no == 1 && do_print)
                        //    printf("S_OFF: %d, Node: Sub, Nums: '%f-%f', Stack: %f \n", S_OFF, var1, var2, stack[S_OFF]);
                    } else if (function == Function::MUL) {
                        stack[S_OFF] = var1 * var2;
                        top++;
                        //if( dataset_no == 1 && do_print)
                        //    printf("S_OFF: %d, Node: Mul, Nums: '%f*%f', Stack: %f \n", S_OFF, var1, var2, stack[S_OFF]);
                    } else if (function == Function::DIV) {
                        if (var2 == 0) {
                            var2 = DELTA;
                        }
                        stack[S_OFF] = var1 / var2;
                        top++;
                        //if( dataset_no == 1 && do_print)
                        //    printf("S_OFF: %d, Node: Div, Nums: '%f/%f', Stack: %f \n", S_OFF, var1, var2, stack[S_OFF]);

                    } else if (function == Function::MAX) {
                        stack[S_OFF] = var1 >= var2 ? var1 : var2;
                        top++;
                    } else if (function == Function::MIN) {
                        stack[S_OFF] = var1 <= var2 ? var1 : var2;
                        top++;
                    }
                }

                if( dataset_no == 1 && do_print)
                    printf("Final ds=1 value %f \n", stack[S_OFF]);
            }

            top--;
            float prefix_value = stack[S_OFF];
            float label_value;
            if(is_subset)
                label_value = label[idxs[dataset_no]];
            else 
                label_value = label[dataset_no];
            float loss = prefix_value - label_value;
            float fitness = loss * loss;

            shared[threadIdx.x] = fitness;

            //if(do_print)
            //    printf("%f,%f,%f,%f, Thread: %d, Shared: %f \n", prefix_value, label_value, loss, fitness, threadIdx.x, shared[threadIdx.x]);

        }

        //printf("Args No Thread: %d, %d, dataset_no: %d,  Thread: %d, Shared: %f  \n", len, dataset_size, dataset_no, threadIdx.x, shared[threadIdx.x]);

        __syncthreads();

        // do parallel reduction
#if THREAD_PER_BLOCK >= 1024
        if (threadIdx.x < 512) { shared[threadIdx.x] += shared[threadIdx.x + 512]; }
        __syncthreads();
#endif
#if THREAD_PER_BLOCK >= 512
        if (threadIdx.x < 256) { shared[threadIdx.x] += shared[threadIdx.x + 256]; }
        __syncthreads();
#endif
        if (threadIdx.x < 128) { shared[threadIdx.x] += shared[threadIdx.x + 128]; }
        __syncthreads();

        if (threadIdx.x < 64) { shared[threadIdx.x] += shared[threadIdx.x + 64]; }
        __syncthreads();

        if (threadIdx.x < 32) { shared[threadIdx.x] += shared[threadIdx.x + 32]; }
        __syncthreads();

        if (threadIdx.x < 16) { shared[threadIdx.x] += shared[threadIdx.x + 16]; }
        __syncthreads();

        if (threadIdx.x < 8) { shared[threadIdx.x] += shared[threadIdx.x + 8]; }
        __syncthreads();

        if (threadIdx.x < 4) { shared[threadIdx.x] += shared[threadIdx.x + 4]; }
        __syncthreads();

        if (threadIdx.x < 2) { shared[threadIdx.x] += shared[threadIdx.x + 2]; }
        __syncthreads();
        
        if (threadIdx.x < 1) {
            
            shared[threadIdx.x] += shared[threadIdx.x + 1];
            result[blockIdx.x] = shared[0];

            //if(do_print)
            //    printf("Final score: %f \n", shared[0]);
        }
    }

    float *mallocStack(int blockNum) {
        float *stack;

        // allocate stack space, the size of which = sizeof(float) * THREAD_PER_BLOCK * (maxDepth + 1)
        hipMalloc((void **) &stack, sizeof(float) * THREAD_PER_BLOCK * (CUSR_DEPTH + 1) * blockNum);

        return stack;
    }

    void calSingleProgram(GPUDataset &dataset, int blockNum, Program &program,
                            float *stack, float *result, float *h_res, metric_t metric) {

        // --------- restrict the length of prefix ---------
        assert(program.length < MAX_PREFIX_LEN);
        // -------------------------------------------------

        // -------- copy to constant memory --------
        float h_nodeValue[MAX_PREFIX_LEN];
        float h_nodeType[MAX_PREFIX_LEN];

        for (int i = 0; i < program.length; i++) {
            int type = program.prefix[i].node_type;
            h_nodeType[i] = type;
            if (type == NodeType::CONST) {
                h_nodeValue[i] = program.prefix[i].constant;
            } else if (type == NodeType::VAR) {
                h_nodeValue[i] = program.prefix[i].variable;
            } else { // unary function or binary function
                h_nodeValue[i] = program.prefix[i].function;
            }
        }

        hipMemcpyToSymbol(HIP_SYMBOL(d_nodeValue), h_nodeValue, sizeof(float) * program.length);
        hipMemcpyToSymbol(HIP_SYMBOL(d_nodeType), h_nodeType, sizeof(float) * program.length);

        int size;

        // -------- calculation and synchronization --------
        //if (metric == metric_t::mean_absolute_error) {
        //    calFitnessGPU_MAE<<<blockNum, THREAD_PER_BLOCK, sizeof(float) * THREAD_PER_BLOCK>>>
        //            (program.length, dataset.dataset, dataset.dataset_pitch, dataset.label, stack, result,
        //                dataset.dataset_size);
        //    hipDeviceSynchronize();
        //} else if (metric == metric_t::mean_square_error || metric == metric_t::root_mean_square_error) {
        if (metric == metric_t::mean_square_error || metric == metric_t::root_mean_square_error) {

            
            if( dataset.subset_size > 0 )
                size = dataset.subset_size;
            else 
                size = dataset.dataset_size;

            calFitnessGPU_MSE<<<blockNum, THREAD_PER_BLOCK, sizeof(float) * THREAD_PER_BLOCK >>>
                (program.length, dataset.dataset, dataset.dataset_pitch, dataset.label, dataset.subset, stack, result,
                    size, dataset.subset_size > 0, false);            
            hipDeviceSynchronize();                

        }

        // -------- reduction on the result --------
        hipMemcpy(h_res, result, sizeof(float) * blockNum, hipMemcpyDeviceToHost);
        float ans = 0;

        for (int i = 0; i < blockNum; i++) {
            ans += h_res[i];
        }

        if (metric == metric_t::mean_absolute_error || metric == metric_t::mean_square_error) {
            program.fitness = ans / (float) size;
        } else if (metric == metric_t::root_mean_square_error) {
            program.fitness = std::sqrt(ans / (float) size);
        }
    }

    float 
    calculateFitness(GPUDataset &dataset, int blockNum, vector<Program> &population, metric_t metric) {

        // allocate space for result
        float *result;
        hipMalloc((void **) &result, sizeof(float) * blockNum);

        // allocate stack space
        float *stack = mallocStack(blockNum);

        // save result and do CPU side reduction
        float *h_res = new float[blockNum];

        // evaluate fitness for each program in the population
        calSingleProgram(dataset, blockNum, population[0], stack, result, h_res, metric);

        // free memory space
        hipFree(result);
        hipFree(stack);
        delete[] h_res;

        return population[0].fitness;
    }

}

